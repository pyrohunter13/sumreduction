#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>
using namespace std;
#define BLOCK_SIZE 16
#define SHMEM_SIZE 256

__global__ void sumReduction(int *a_d, int *b_d) {
	extern __shared__ int sdata[SHMEM_SIZE];
	int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[threadIdx.x] = a_d[i];
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
        sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
        }
	if (threadIdx.x == 0) b_d[blockIdx.x] = sdata[0];
}

int main() {
	int n = 10;
    // size_t bytes = n * sizeof(int);
    srand(time(0));
    // initiate host
    // vector<int> a_h(n);
	// vector<int> b_h(n);
    // generate(begin(a_h), end(a_h), [](){ return rand() % 10; });
	int *a_h, *b_h;
    a_h = (int *) malloc(n*sizeof(int));
    b_h = (int *) malloc(n*sizeof(int));
    for(int i=0;i<n;i++)a_h[i]=1;
    
    // initiate device
    // int *a_d, *b_d;
	// hipMalloc(&a_d, bytes);
	// hipMalloc(&b_d, bytes);
    int *a_d,*b_d;
    hipMalloc((void **) &a_d, n*sizeof(int));
	hipMalloc((void **) &b_d, n*sizeof(int));
    
    //data movement
    // hipMemcpy(a_d,a_d.data(),bytes, hipMemcpyHostToDevice);
    hipMemcpy(a_d,a_h,n*sizeof(int),hipMemcpyHostToDevice);
    
    //initiate grid and block size
    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    //call kernel
    sumReduction<<<dimGrid,dimBlock>>>(a_d, b_d);

    //barrier
    
    //data movement
    hipMemcpy(a_h,a_d,n*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(b_h,b_d,n*sizeof(int),hipMemcpyDeviceToHost);
    
    //hasil
    for(int i=0;i<n;i++)printf("%d ", a_h[i]);
    printf("\n");
    printf("Sum: %d", b_h[0]);

    hipFree(a_d);
    hipFree(b_h);
    free(a_h);
    free(b_h);

	return 0;
}